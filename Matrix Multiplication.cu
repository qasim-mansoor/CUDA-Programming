#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

int main(int argc, char const *argv[])
{
    int m = 4, n = 4, k = 4;

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = i * n + j;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = (i + n) * n + j;
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    // Launch kernel 
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k); 
    
    // Transefr results from device to host 
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    

    // validate results computed by GPU
    printf("Matrix A: \n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("%.2d ",  h_a[i * k + j]);
        }
        printf("\n");
    }
 
     printf("\n");
 
    printf("Matrix B: \n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("%.2d ",  h_b[i * k + j]);
        }
        printf("\n");
    }
 
    printf("\n");

    printf("Matrix C: \n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("%.4d ",  h_c[i * k + j]);
        }
        printf("\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}